#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
 #include <hip/hip_runtime.h>

#define STB_IMAGE_IMPLEMENTATION
#include "../../include/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../../include/stb_image_write.h"


__global__ void horizontal_flip_image(unsigned char* horiz_flip_img,unsigned char* img,int height,int width)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;


	if(row < height && col < width )
	{	
		horiz_flip_img[3 * (col * width + row) + 0] = img[3 * (col * width + (height-1-row)) + 0];
		horiz_flip_img[3 * (col * width + row) + 1] = img[3 * (col * width + (height-1-row)) + 1];
		horiz_flip_img[3 * (col * width + row) + 2] = img[3 * (col * width + (height-1-row)) + 2];

	}
}


 
int main(void) {
	int width, height, channels;
	int original_channels;
	const char *fname = "high_res.jpg";
	stbi_info(fname, &width, &height, &channels);

	unsigned char *img = stbi_load(fname, &width, &height, &original_channels, channels);
	if(img == NULL) {
		printf("Error in loading the image\n");
		exit(1);
	}
	printf("Loaded image with a width of %dpx, a height of %dpx , %d Originals channels and %d channels\n", width, height, original_channels, channels);

	
	size_t img_size = width * height * channels;

	unsigned char *horiz_flip_img = (unsigned char*)malloc(img_size);
	if(horiz_flip_img == NULL) {
		printf("Unable to allocate memory for the rotate image.\n");
		exit(1);
	}	


	unsigned char *d_img =NULL;
	unsigned char *d_horiz_flip_img=NULL;
	
	hipMalloc(( void **)&d_img, sizeof(unsigned char)*img_size);
	hipMalloc(( void **)&d_horiz_flip_img, sizeof(unsigned char)*img_size);

	hipMemcpy(d_img, img, sizeof(unsigned char)*img_size, hipMemcpyHostToDevice);
	hipMemcpy(d_horiz_flip_img, horiz_flip_img, sizeof(unsigned char)*img_size, hipMemcpyHostToDevice);
	
	int NUM_THREADS = 4;
	dim3 threadsPerBlock(NUM_THREADS, NUM_THREADS);

	dim3 blocksPerGrid(ceil(double(width/NUM_THREADS)), ceil(double(height/NUM_THREADS)));
	horizontal_flip_image<<<blocksPerGrid,threadsPerBlock>>>(d_horiz_flip_img,d_img,height,width);
	
	hipDeviceSynchronize();

	hipMemcpy(horiz_flip_img, d_horiz_flip_img, sizeof(unsigned char)*img_size, hipMemcpyDeviceToHost);

	hipFree(d_img);
	hipFree(d_horiz_flip_img);
	stbi_write_jpg("horizontal_flipped_img_cuda.png", width, height, channels, horiz_flip_img, 100);

}
