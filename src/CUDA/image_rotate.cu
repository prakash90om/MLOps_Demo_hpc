#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>



#define STB_IMAGE_IMPLEMENTATION
#include "../../include/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../../include/stb_image_write.h"


__global__ void image_rotate(unsigned char* rotate_img,unsigned char* img,int height,int width)
{
		int col = threadIdx.x + blockDim.x * blockIdx.x;
		int row = threadIdx.y + blockDim.y * blockIdx.y;

    double degree =20.0;
    double radians = (degree * 3.1415926)/180.0;
    double sin_rad = sin(-radians);
    double cos_rad = cos(-radians);

    double row_center = 0.5*(height);
    double col_center = 0.5*(width);

    double a = row - row_center;
    double b = col - col_center ;

    int row_orig = (int)(a * cos_rad - b*sin_rad + row_center);
    int col_orig = (int)(a * sin_rad + b*cos_rad + col_center);

		if(row < height && col < width )
		{	
      if(row_orig >=0 && row_orig <height && col_orig>=0 && col_orig < width)
      {
        rotate_img[3 * (row * width + col) + 0] = img[3 * (row_orig * width + col_orig) + 0];
        rotate_img[3 * (row * width + col) + 1] = img[3 * (row_orig * width + col_orig) + 1];
        rotate_img[3 * (row * width + col) + 2] = img[3 * (row_orig * width + col_orig) + 2];
      }
		}
}
 
int main(void) {
	int width, height, channels;
	int original_channels;
	const char *fname = "high_res_images/high_res.jpg";
	stbi_info(fname, &width, &height, &channels);

	unsigned char *img = stbi_load(fname, &width, &height, &original_channels, channels);
	if(img == NULL) {
		printf("Error in loading the image\n");
		exit(1);
	}
	printf("Loaded image with a width of %dpx, a height of %dpx , %d Originals channels and %d channels\n", width, height, original_channels, channels);

	
	size_t img_size = width * height * channels;

	unsigned char *rotate_img = (unsigned char*)malloc(img_size);
	if(rotate_img == NULL) {
		printf("Unable to allocate memory for the rotate image.\n");
		exit(1);
	}	


	unsigned char *d_img =NULL;
	unsigned char *d_rotate_img=NULL;
	
  hipMalloc(( void **)&d_img, sizeof(unsigned char)*img_size);
  hipMalloc(( void **)&d_rotate_img, sizeof(unsigned char)*img_size);

  hipMemcpy(d_img, img, sizeof(unsigned char)*img_size, hipMemcpyHostToDevice);
  hipMemcpy(d_rotate_img, rotate_img, sizeof(unsigned char)*img_size, hipMemcpyHostToDevice);
 
  int NUM_THREADS = 32;
  dim3 threadsPerBlock(NUM_THREADS, NUM_THREADS);

  dim3 blocksPerGrid(ceil(double(width/NUM_THREADS)), ceil(double(height/NUM_THREADS)));
  struct timespec start, end;
  clock_gettime(CLOCK_MONOTONIC, &start);
  image_rotate<<<blocksPerGrid,threadsPerBlock>>>(d_rotate_img,d_img,height,width);
  
  hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &end);
    double time_taken;
    time_taken = (end.tv_sec - start.tv_sec) * 1e9;
    time_taken = (time_taken + (end.tv_nsec - start.tv_nsec)) * 1e-9;
  
    printf("Time taken - Image Rotate: %f\n",time_taken);
  hipMemcpy(rotate_img, d_rotate_img, sizeof(unsigned char)*img_size, hipMemcpyDeviceToHost);

  hipFree(d_img);
  hipFree(d_rotate_img);
  stbi_write_jpg("augmented_images/image_rotate.jpg", width, height, channels, rotate_img, 100);

}
